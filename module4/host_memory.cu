/******************************************************************************
 * host_memory.cu
 *
 * Based on the original SAXPY sample from NVIDIA's devblogs and our earlier
 * experiments, this version performs SAXPY operations over a range of vector
 * sizes, timing both GPU and CPU executions and reporting results in CSV format.
 *
 * Additional timing measurements have been added to help analyze performance.
 ******************************************************************************/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>

// -----------------------------------------------------------------------------
// Error-checking macro
// -----------------------------------------------------------------------------
#define CUDA_CHECK(call) do {                                      \
    hipError_t err = call;                                        \
    if(err != hipSuccess){                                        \
        fprintf(stderr, "CUDA error in %s at line %d: %s\n",         \
                __FILE__, __LINE__, hipGetErrorString(err));      \
        exit(err);                                                 \
    }                                                              \
} while(0)

// -----------------------------------------------------------------------------
// SAXPY kernel (as in the original)
// -----------------------------------------------------------------------------
__global__
void saxpy(int n, float a, float *x, float *y) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        y[i] = a * x[i] + y[i];
    }
}

// -----------------------------------------------------------------------------
// CPU SAXPY (reference implementation)
// -----------------------------------------------------------------------------
void saxpy_cpu(int n, float a, const float *x, float *y) {
    for (int i = 0; i < n; i++) {
        y[i] = a * x[i] + y[i];
    }
}

// -----------------------------------------------------------------------------
// Timing helpers using CUDA events
// -----------------------------------------------------------------------------
static inline hipEvent_t get_time(void) {
    hipEvent_t ev;
    CUDA_CHECK(hipEventCreate(&ev));
    CUDA_CHECK(hipEventRecord(ev, 0));
    CUDA_CHECK(hipEventSynchronize(ev));
    return ev;
}

static inline float elapsed_time(hipEvent_t start, hipEvent_t end) {
    float ms = 0.0f;
    CUDA_CHECK(hipEventElapsedTime(&ms, start, end));
    return ms;
}

// -----------------------------------------------------------------------------
// Main: Run SAXPY experiments over various vector sizes
// -----------------------------------------------------------------------------
int main(void) {
    printf("=== SAXPY Experiments ===\n");
    printf("CSV Format: VectorSize,GPUSaxpyTime_ms,CPUSaxpyTime_ms,MaxError\n");

    // Test vector sizes from 2^16 to 2^23 (65,536 to ~8 million elements)
    int sizes[] = {1<<16, 1<<17, 1<<18, 1<<19, 1<<20, 1<<21, 1<<22, 1<<23};
    int num_sizes = sizeof(sizes) / sizeof(sizes[0]);
    float a = 2.0f;
    for (int s = 0; s < num_sizes; s++) {
        int N = sizes[s];
        float *h_x = (float*) malloc(N * sizeof(float));
        float *h_y = (float*) malloc(N * sizeof(float));
        float *h_y_cpu = (float*) malloc(N * sizeof(float));

        // Initialize input data
        for (int i = 0; i < N; i++) {
            h_x[i] = 1.0f;
            h_y[i] = 2.0f;
            h_y_cpu[i] = 2.0f;
        }

        // Allocate device memory
        float *d_x = NULL, *d_y = NULL;
        CUDA_CHECK(hipMalloc(&d_x, N * sizeof(float)));
        CUDA_CHECK(hipMalloc(&d_y, N * sizeof(float)));

        CUDA_CHECK(hipMemcpy(d_x, h_x, N * sizeof(float), hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(d_y, h_y, N * sizeof(float), hipMemcpyHostToDevice));

        // GPU SAXPY timing
        dim3 block(256);
        dim3 grid((N + block.x - 1) / block.x);
        hipEvent_t start_gpu = get_time();
        saxpy<<<grid, block>>>(N, a, d_x, d_y);
        CUDA_CHECK(hipDeviceSynchronize());
        hipEvent_t end_gpu = get_time();
        float gpuTime = elapsed_time(start_gpu, end_gpu);

        // Copy GPU result back to host
        CUDA_CHECK(hipMemcpy(h_y, d_y, N * sizeof(float), hipMemcpyDeviceToHost));

        // CPU SAXPY timing
        hipEvent_t start_cpu = get_time();
        saxpy_cpu(N, a, h_x, h_y_cpu);
        hipEvent_t end_cpu = get_time();
        float cpuTime = elapsed_time(start_cpu, end_cpu);

        // Calculate maximum error between GPU and CPU results
        float maxError = 0.0f;
        for (int i = 0; i < N; i++) {
            float err = fabs(h_y[i] - h_y_cpu[i]);
            if (err > maxError)
                maxError = err;
        }
        printf("%d,%.4f,%.4f,%.5f\n", N, gpuTime, cpuTime, maxError);

        free(h_x);
        free(h_y);
        free(h_y_cpu);
        CUDA_CHECK(hipFree(d_x));
        CUDA_CHECK(hipFree(d_y));
    }
    CUDA_CHECK(hipDeviceReset());
    return 0;
}

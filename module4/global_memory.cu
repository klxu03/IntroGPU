/******************************************************************************
 * global_memory_experiments.cu
 *
 * Demonstrates interleaved vs. non-interleaved array access on CPU and GPU
 * for various array sizes and iteration counts, printing out timing results
 * in CSV format. Also includes a simple bitreverse kernel test with multiple
 * block sizes for more experiments.
 ******************************************************************************/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <assert.h>

// -----------------------------------------------------------------------------
// Error-checking macro
// -----------------------------------------------------------------------------
#define CUDA_CHECK(call) do {                                      \
    hipError_t err = call;                                        \
    if (err != hipSuccess) {                                      \
        fprintf(stderr, "CUDA error in %s at line %d: %s\n",       \
                __FILE__, __LINE__, hipGetErrorString(err));      \
        exit(err);                                                 \
    }                                                              \
} while(0)

// -----------------------------------------------------------------------------
// Timing helpers
// -----------------------------------------------------------------------------
static inline hipEvent_t get_time(void)
{
    hipEvent_t time;
    hipEventCreate(&time);
    hipEventRecord(time, 0);
    hipEventSynchronize(time);
    return time;
}

static inline float elapsed_time(hipEvent_t start, hipEvent_t end)
{
    float ms = 0.0f;
    hipEventElapsedTime(&ms, start, end);
    return ms;
}

// -----------------------------------------------------------------------------
// Interleaved vs. Non-interleaved Data Structures
// -----------------------------------------------------------------------------
typedef struct {
    unsigned int a;
    unsigned int b;
    unsigned int c;
    unsigned int d;
} INTERLEAVED_T;

// Non-interleaved: 4 separate arrays
typedef struct {
    unsigned int *a;
    unsigned int *b;
    unsigned int *c;
    unsigned int *d;
} NON_INTERLEAVED_T;

// -----------------------------------------------------------------------------
// CPU addition: Interleaved
// -----------------------------------------------------------------------------
float cpu_add_interleaved(INTERLEAVED_T *dest, const INTERLEAVED_T *src,
                          unsigned int num_elements, unsigned int iter)
{
    hipEvent_t start = get_time();
    for (unsigned int i = 0; i < num_elements; i++) {
        for (unsigned int j = 0; j < iter; j++) {
            dest[i].a += src[i].a;
            dest[i].b += src[i].b;
            dest[i].c += src[i].c;
            dest[i].d += src[i].d;
        }
    }
    hipEvent_t end = get_time();
    return elapsed_time(start, end);
}

// -----------------------------------------------------------------------------
// CPU addition: Non-interleaved
// -----------------------------------------------------------------------------
float cpu_add_noninterleaved(NON_INTERLEAVED_T &dest, const NON_INTERLEAVED_T &src,
                             unsigned int num_elements, unsigned int iter)
{
    hipEvent_t start = get_time();
    for (unsigned int i = 0; i < num_elements; i++) {
        for (unsigned int j = 0; j < iter; j++) {
            dest.a[i] += src.a[i];
            dest.b[i] += src.b[i];
            dest.c[i] += src.c[i];
            dest.d[i] += src.d[i];
        }
    }
    hipEvent_t end = get_time();
    return elapsed_time(start, end);
}

// -----------------------------------------------------------------------------
// GPU kernel: Interleaved
// -----------------------------------------------------------------------------
__global__ void add_kernel_interleaved(INTERLEAVED_T *dest, const INTERLEAVED_T *src,
                                       unsigned int num_elements, unsigned int iter)
{
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < num_elements) {
        for (unsigned int i = 0; i < iter; i++) {
            dest[tid].a += src[tid].a;
            dest[tid].b += src[tid].b;
            dest[tid].c += src[tid].c;
            dest[tid].d += src[tid].d;
        }
    }
}

// -----------------------------------------------------------------------------
// GPU kernel: Non-interleaved
// -----------------------------------------------------------------------------
__global__ void add_kernel_noninterleaved(NON_INTERLEAVED_T dest, NON_INTERLEAVED_T src,
                                          unsigned int num_elements, unsigned int iter)
{
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < num_elements) {
        for (unsigned int i = 0; i < iter; i++) {
            dest.a[tid] += src.a[tid];
            dest.b[tid] += src.b[tid];
            dest.c[tid] += src.c[tid];
            dest.d[tid] += src.d[tid];
        }
    }
}

// -----------------------------------------------------------------------------
// GPU addition: Interleaved
// -----------------------------------------------------------------------------
float gpu_add_interleaved(INTERLEAVED_T *h_dest, const INTERLEAVED_T *h_src,
                          unsigned int num_elements, unsigned int iter)
{
    // Allocate device memory
    size_t bytes = num_elements * sizeof(INTERLEAVED_T);
    INTERLEAVED_T *d_dest = nullptr, *d_src = nullptr;
    CUDA_CHECK(hipMalloc(&d_dest, bytes));
    CUDA_CHECK(hipMalloc(&d_src,  bytes));

    // Copy data to device
    CUDA_CHECK(hipMemcpy(d_src, h_src, bytes, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemset(d_dest, 0, bytes)); // start from zero to see the addition cost

    // Launch kernel
    dim3 block(256);
    dim3 grid((num_elements + block.x - 1) / block.x);

    hipEvent_t start = get_time();
    add_kernel_interleaved<<<grid, block>>>(d_dest, d_src, num_elements, iter);
    hipEvent_t end = get_time();
    CUDA_CHECK(hipDeviceSynchronize()); // ensure kernel finishes

    float ms = elapsed_time(start, end);

    // Copy results back (optional if you just want the timing)
    CUDA_CHECK(hipMemcpy(h_dest, d_dest, bytes, hipMemcpyDeviceToHost));

    // Clean up
    CUDA_CHECK(hipFree(d_dest));
    CUDA_CHECK(hipFree(d_src));
    return ms;
}

// -----------------------------------------------------------------------------
// GPU addition: Non-interleaved
// -----------------------------------------------------------------------------
float gpu_add_noninterleaved(NON_INTERLEAVED_T &h_dest, const NON_INTERLEAVED_T &h_src,
                             unsigned int num_elements, unsigned int iter)
{
    // Allocate device memory
    size_t bytes = num_elements * sizeof(unsigned int);
    NON_INTERLEAVED_T d_dest, d_src;
    CUDA_CHECK(hipMalloc(&d_dest.a, bytes));
    CUDA_CHECK(hipMalloc(&d_dest.b, bytes));
    CUDA_CHECK(hipMalloc(&d_dest.c, bytes));
    CUDA_CHECK(hipMalloc(&d_dest.d, bytes));

    CUDA_CHECK(hipMalloc(&d_src.a, bytes));
    CUDA_CHECK(hipMalloc(&d_src.b, bytes));
    CUDA_CHECK(hipMalloc(&d_src.c, bytes));
    CUDA_CHECK(hipMalloc(&d_src.d, bytes));

    // Copy data to device
    CUDA_CHECK(hipMemcpy(d_src.a, h_src.a, bytes, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_src.b, h_src.b, bytes, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_src.c, h_src.c, bytes, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_src.d, h_src.d, bytes, hipMemcpyHostToDevice));

    // Zero out dest
    CUDA_CHECK(hipMemset(d_dest.a, 0, bytes));
    CUDA_CHECK(hipMemset(d_dest.b, 0, bytes));
    CUDA_CHECK(hipMemset(d_dest.c, 0, bytes));
    CUDA_CHECK(hipMemset(d_dest.d, 0, bytes));

    // Launch kernel
    dim3 block(256);
    dim3 grid((num_elements + block.x - 1) / block.x);

    hipEvent_t start = get_time();
    add_kernel_noninterleaved<<<grid, block>>>(d_dest, d_src, num_elements, iter);
    hipEvent_t end = get_time();
    CUDA_CHECK(hipDeviceSynchronize());

    float ms = elapsed_time(start, end);

    // Copy results back
    CUDA_CHECK(hipMemcpy(h_dest.a, d_dest.a, bytes, hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(h_dest.b, d_dest.b, bytes, hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(h_dest.c, d_dest.c, bytes, hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(h_dest.d, d_dest.d, bytes, hipMemcpyDeviceToHost));

    // Clean up
    CUDA_CHECK(hipFree(d_dest.a));
    CUDA_CHECK(hipFree(d_dest.b));
    CUDA_CHECK(hipFree(d_dest.c));
    CUDA_CHECK(hipFree(d_dest.d));
    CUDA_CHECK(hipFree(d_src.a));
    CUDA_CHECK(hipFree(d_src.b));
    CUDA_CHECK(hipFree(d_src.c));
    CUDA_CHECK(hipFree(d_src.d));

    return ms;
}

// -----------------------------------------------------------------------------
// Simple bitreverse kernel for extra experiments
// -----------------------------------------------------------------------------
__host__ __device__ unsigned int bitreverse_func(unsigned int number) {
    number = ((0xf0f0f0f0 & number) >> 4)  | ((0x0f0f0f0f & number) << 4);
    number = ((0xcccccccc & number) >> 2)  | ((0x33333333 & number) << 2);
    number = ((0xaaaaaaaa & number) >> 1)  | ((0x55555555 & number) << 1);
    return number;
}

__global__ void bitreverse_kernel(unsigned int *data, unsigned int size)
{
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < size) {
        data[tid] = bitreverse_func(data[tid]);
    }
}

// -----------------------------------------------------------------------------
// Main
// -----------------------------------------------------------------------------
int main()
{
    printf("=== Interleaved vs Non-Interleaved Memory Experiments ===\n");
    printf("CSV Format: size,iter,cpu_interleaved_ms,gpu_interleaved_ms,cpu_noninterleaved_ms,gpu_noninterleaved_ms\n");

    // We'll test array sizes from 256 to 131072, doubling each time.
    // We'll test iteration counts from 1, 2, 4, 8, 16, 32.
    unsigned int sizes[] = {256, 512, 1024, 2048, 4096, 8192, 16384, 32768, 65536, 131072};
    unsigned int iters[] = {1, 2, 4, 8, 16, 32};

    for (unsigned int s = 0; s < sizeof(sizes)/sizeof(sizes[0]); s++) {
        unsigned int num_elements = sizes[s];

        // Allocate host memory for interleaved
        INTERLEAVED_T *host_src_i = (INTERLEAVED_T*) malloc(num_elements * sizeof(INTERLEAVED_T));
        INTERLEAVED_T *host_dest_i = (INTERLEAVED_T*) malloc(num_elements * sizeof(INTERLEAVED_T));

        // Allocate host memory for non-interleaved
        NON_INTERLEAVED_T host_src_n, host_dest_n;
        host_src_n.a = (unsigned int*) malloc(num_elements * sizeof(unsigned int));
        host_src_n.b = (unsigned int*) malloc(num_elements * sizeof(unsigned int));
        host_src_n.c = (unsigned int*) malloc(num_elements * sizeof(unsigned int));
        host_src_n.d = (unsigned int*) malloc(num_elements * sizeof(unsigned int));

        host_dest_n.a = (unsigned int*) malloc(num_elements * sizeof(unsigned int));
        host_dest_n.b = (unsigned int*) malloc(num_elements * sizeof(unsigned int));
        host_dest_n.c = (unsigned int*) malloc(num_elements * sizeof(unsigned int));
        host_dest_n.d = (unsigned int*) malloc(num_elements * sizeof(unsigned int));

        // Initialize data
        for (unsigned int i = 0; i < num_elements; i++) {
            host_src_i[i].a = i;
            host_src_i[i].b = i + 1;
            host_src_i[i].c = i + 2;
            host_src_i[i].d = i + 3;

            // mirror data for non-interleaved
            host_src_n.a[i] = i;
            host_src_n.b[i] = i + 1;
            host_src_n.c[i] = i + 2;
            host_src_n.d[i] = i + 3;

            // zero out dest
            host_dest_i[i].a = 0;
            host_dest_i[i].b = 0;
            host_dest_i[i].c = 0;
            host_dest_i[i].d = 0;

            host_dest_n.a[i] = 0;
            host_dest_n.b[i] = 0;
            host_dest_n.c[i] = 0;
            host_dest_n.d[i] = 0;
        }

        // Now loop over iteration counts
        for (unsigned int it = 0; it < sizeof(iters)/sizeof(iters[0]); it++) {
            unsigned int iter = iters[it];

            // CPU times
            float cpu_int_ms = cpu_add_interleaved(host_dest_i, host_src_i, num_elements, iter);

            // Re-zero out host_dest_i for a fair GPU test
            for (unsigned int i = 0; i < num_elements; i++) {
                host_dest_i[i].a = 0;
                host_dest_i[i].b = 0;
                host_dest_i[i].c = 0;
                host_dest_i[i].d = 0;
            }

            float gpu_int_ms = gpu_add_interleaved(host_dest_i, host_src_i, num_elements, iter);

            // CPU times (non-interleaved)
            float cpu_nint_ms = cpu_add_noninterleaved(host_dest_n, host_src_n, num_elements, iter);

            // Re-zero out host_dest_n for GPU
            for (unsigned int i = 0; i < num_elements; i++) {
                host_dest_n.a[i] = 0;
                host_dest_n.b[i] = 0;
                host_dest_n.c[i] = 0;
                host_dest_n.d[i] = 0;
            }

            float gpu_nint_ms = gpu_add_noninterleaved(host_dest_n, host_src_n, num_elements, iter);

            // Print CSV line
            printf("%u,%u,%.4f,%.4f,%.4f,%.4f\n",
                   num_elements, iter, cpu_int_ms, gpu_int_ms, cpu_nint_ms, gpu_nint_ms);
        }

        // Free memory
        free(host_src_i);
        free(host_dest_i);

        free(host_src_n.a);  free(host_src_n.b);
        free(host_src_n.c);  free(host_src_n.d);
        free(host_dest_n.a); free(host_dest_n.b);
        free(host_dest_n.c); free(host_dest_n.d);
    }

    // -------------------------------------------------------------------------
    // Extra experiment: bitreverse with different block sizes
    // -------------------------------------------------------------------------
    printf("\n=== Bitreverse Experiments ===\n");
    printf("CSV Format: arraySize,blockSize,bitreverseTime(ms)\n");

    // Let’s pick an array size for bitreverse
    const unsigned int BITREVERSE_SIZE = 1 << 16; // 65536
    unsigned int *h_data = (unsigned int*) malloc(BITREVERSE_SIZE * sizeof(unsigned int));
    for (unsigned int i = 0; i < BITREVERSE_SIZE; i++) {
        h_data[i] = i;
    }

    // Copy to device once, then do multiple block sizes
    unsigned int *d_data = nullptr;
    CUDA_CHECK(hipMalloc(&d_data, BITREVERSE_SIZE * sizeof(unsigned int)));
    CUDA_CHECK(hipMemcpy(d_data, h_data, BITREVERSE_SIZE * sizeof(unsigned int),
                          hipMemcpyHostToDevice));

    // Test block sizes from 64 to 1024
    for (unsigned int blockSize = 64; blockSize <= 1024; blockSize *= 2) {
        dim3 block(blockSize);
        dim3 grid((BITREVERSE_SIZE + blockSize - 1) / blockSize);

        // Re-initialize device data each time so timing is consistent
        CUDA_CHECK(hipMemcpy(d_data, h_data, BITREVERSE_SIZE * sizeof(unsigned int),
                              hipMemcpyHostToDevice));

        hipEvent_t start = get_time();
        bitreverse_kernel<<<grid, block>>>(d_data, BITREVERSE_SIZE);
        CUDA_CHECK(hipDeviceSynchronize());
        hipEvent_t end = get_time();

        float ms = elapsed_time(start, end);
        printf("%u,%u,%.4f\n", BITREVERSE_SIZE, blockSize, ms);
    }

    CUDA_CHECK(hipFree(d_data));
    free(h_data);

    CUDA_CHECK(hipDeviceReset());
    return 0;
}
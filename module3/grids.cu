#include <stdio.h>
#include <hip/hip_runtime.h>

// Kernel (unchanged)
__global__ void what_is_my_id_2d_A(
    unsigned int * const block_x,
    unsigned int * const block_y,
    unsigned int * const thread,
    unsigned int * const calc_thread,
    unsigned int * const x_thread,
    unsigned int * const y_thread,
    unsigned int * const grid_dimx,
    unsigned int * const block_dimx,
    unsigned int * const grid_dimy,
    unsigned int * const block_dimy)
{
    const unsigned int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    const unsigned int idy = (blockIdx.y * blockDim.y) + threadIdx.y;
    const unsigned int thread_idx = ((gridDim.x * blockDim.x) * idy) + idx;

    block_x[thread_idx]    = blockIdx.x;
    block_y[thread_idx]    = blockIdx.y;
    thread[thread_idx]     = threadIdx.x;
    calc_thread[thread_idx]= thread_idx;
    x_thread[thread_idx]   = idx;
    y_thread[thread_idx]   = idy;
    grid_dimx[thread_idx]  = gridDim.x;
    block_dimx[thread_idx] = blockDim.x;
    grid_dimy[thread_idx]  = gridDim.y;
    block_dimy[thread_idx] = blockDim.y;
}

// Increase problem size for timing tests
#define ARRAY_SIZE_X 256
#define ARRAY_SIZE_Y 256
#define ARRAY_SIZE_IN_BYTES ((ARRAY_SIZE_X) * (ARRAY_SIZE_Y) * (sizeof(unsigned int)))

// Declare CPU arrays (statically, for demonstration)
unsigned int cpu_block_x[ARRAY_SIZE_Y][ARRAY_SIZE_X];
unsigned int cpu_block_y[ARRAY_SIZE_Y][ARRAY_SIZE_X];
unsigned int cpu_thread[ARRAY_SIZE_Y][ARRAY_SIZE_X];
unsigned int cpu_calc_thread[ARRAY_SIZE_Y][ARRAY_SIZE_X];
unsigned int cpu_xthread[ARRAY_SIZE_Y][ARRAY_SIZE_X];
unsigned int cpu_ythread[ARRAY_SIZE_Y][ARRAY_SIZE_X];
unsigned int cpu_grid_dimx[ARRAY_SIZE_Y][ARRAY_SIZE_X];
unsigned int cpu_block_dimx[ARRAY_SIZE_Y][ARRAY_SIZE_X];
unsigned int cpu_grid_dimy[ARRAY_SIZE_Y][ARRAY_SIZE_X];
unsigned int cpu_block_dimy[ARRAY_SIZE_Y][ARRAY_SIZE_X];

int main(void)
{
    // --- Define three grid/block configurations ---
    // Configuration A: 32x4 threads per block; grid: 1x4 blocks
    const dim3 threads_rect(32, 4);
    const dim3 blocks_rect(1, 4);
    
    // Configuration B: 16x8 threads per block; grid: 2x2 blocks
    const dim3 threads_square(16, 8);
    const dim3 blocks_square(2, 2);
    
    // Configuration C: 16x16 threads per block; grid: (256/16)x(256/16) = 16x16 blocks
    const dim3 threads_16x16(16, 16);
    const dim3 blocks_16x16(ARRAY_SIZE_X / threads_16x16.x, ARRAY_SIZE_Y / threads_16x16.y);

    // --- Create CUDA events for timing ---
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Allocate GPU memory
    unsigned int * gpu_block_x, * gpu_block_y, * gpu_thread, * gpu_calc_thread;
    unsigned int * gpu_xthread, * gpu_ythread, * gpu_grid_dimx, * gpu_block_dimx;
    unsigned int * gpu_grid_dimy, * gpu_block_dimy;
    
    hipMalloc((void **)&gpu_block_x, ARRAY_SIZE_IN_BYTES);
    hipMalloc((void **)&gpu_block_y, ARRAY_SIZE_IN_BYTES);
    hipMalloc((void **)&gpu_thread,  ARRAY_SIZE_IN_BYTES);
    hipMalloc((void **)&gpu_calc_thread, ARRAY_SIZE_IN_BYTES);
    hipMalloc((void **)&gpu_xthread, ARRAY_SIZE_IN_BYTES);
    hipMalloc((void **)&gpu_ythread, ARRAY_SIZE_IN_BYTES);
    hipMalloc((void **)&gpu_grid_dimx, ARRAY_SIZE_IN_BYTES);
    hipMalloc((void **)&gpu_block_dimx, ARRAY_SIZE_IN_BYTES);
    hipMalloc((void **)&gpu_grid_dimy, ARRAY_SIZE_IN_BYTES);
    hipMalloc((void **)&gpu_block_dimy, ARRAY_SIZE_IN_BYTES);

    // Loop over the three configurations (kernel index 0,1,2)
    for(int kernel = 0; kernel < 3; kernel++)
    {
        float totalTime = 0.0f;
        // Run 10 iterations for each configuration
        for (int iter = 0; iter < 10; iter++)
        {
            hipEventRecord(start, 0);
            switch(kernel)
            {
                case 0:
                    // Configuration A
                    what_is_my_id_2d_A<<<blocks_rect, threads_rect>>>(gpu_block_x, gpu_block_y,
                        gpu_thread, gpu_calc_thread, gpu_xthread, gpu_ythread,
                        gpu_grid_dimx, gpu_block_dimx, gpu_grid_dimy, gpu_block_dimy);
                    break;
                case 1:
                    // Configuration B
                    what_is_my_id_2d_A<<<blocks_square, threads_square>>>(gpu_block_x, gpu_block_y,
                        gpu_thread, gpu_calc_thread, gpu_xthread, gpu_ythread,
                        gpu_grid_dimx, gpu_block_dimx, gpu_grid_dimy, gpu_block_dimy);
                    break;
                case 2:
                    // Configuration C
                    what_is_my_id_2d_A<<<blocks_16x16, threads_16x16>>>(gpu_block_x, gpu_block_y,
                        gpu_thread, gpu_calc_thread, gpu_xthread, gpu_ythread,
                        gpu_grid_dimx, gpu_block_dimx, gpu_grid_dimy, gpu_block_dimy);
                    break;
                default:
                    exit(1);
            }
            hipEventRecord(stop, 0);
            hipEventSynchronize(stop);
            
            float elapsedTime = 0.0f;
            hipEventElapsedTime(&elapsedTime, start, stop);
            totalTime += elapsedTime;
        }
        
        // Compute average time over 10 iterations
        float averageTime = totalTime / 10.0f;
        
        // Copy results back from GPU (from the final iteration)
        hipMemcpy(cpu_block_x, gpu_block_x, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
        hipMemcpy(cpu_block_y, gpu_block_y, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
        hipMemcpy(cpu_thread, gpu_thread, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
        hipMemcpy(cpu_calc_thread, gpu_calc_thread, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
        hipMemcpy(cpu_xthread, gpu_xthread, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
        hipMemcpy(cpu_ythread, gpu_ythread, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
        hipMemcpy(cpu_grid_dimx, gpu_grid_dimx, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
        hipMemcpy(cpu_block_dimx, gpu_block_dimx, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
        hipMemcpy(cpu_grid_dimy, gpu_grid_dimy, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
        hipMemcpy(cpu_block_dimy, gpu_block_dimy, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
        
        printf("\nKernel %d average execution time over 10 iterations: %f ms\n", kernel, averageTime);
        printf("First element details: CT: %2u BKX: %1u BKY: %1u TID: %2u XTID: %2u YTID: %2u\n",
               cpu_calc_thread[0][0], cpu_block_x[0][0], cpu_block_y[0][0],
               cpu_thread[0][0], cpu_xthread[0][0], cpu_ythread[0][0]);
    }
    
    // Free GPU memory and destroy events
    hipFree(gpu_block_x);
    hipFree(gpu_block_y);
    hipFree(gpu_thread);
    hipFree(gpu_calc_thread);
    hipFree(gpu_xthread);
    hipFree(gpu_ythread);
    hipFree(gpu_grid_dimx);
    hipFree(gpu_block_dimx);
    hipFree(gpu_grid_dimy);
    hipFree(gpu_block_dimy);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    
    return 0;
}
